#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"


__global__ void
DecayAmp_kernel( GPU_AMP_PROTO, GDouble par_A, GDouble par_B, GDouble par_C, GDouble par_D, GDouble par_E, GDouble par_F, GDouble par_G, GDouble par_H, GDouble par_L){

  int iEvent = GPU_THIS_EVENT;

  /****
  *  Equivalently one could switch needsUserVarsOnly to false
  *  and uncomment the following lines to get the same
  *  result.  The role of user-data is to optimize this function
  *  call in the instance it is repeated multiple times throughout
  *  a fit.

  GDouble p1[4] = GPU_P4(daught1-1);
  GDouble p2[4] = GPU_P4(daught2-1);

  GDouble m = G_SQRT(SQ(p1[0]+p2[0]) - SQ(p1[1]+p2[1])
                                     - SQ(p1[2]+p2[2])
                                     - SQ(p1[3]+p2[3]));

  WCUComplex bwBot = { SQ( m ) - SQ( mass0 ), mass0 * width0 };
  */


  // here we need to be careful to index the user-defined
  // data with the proper integer corresponding to the
  // enumeration in the C++ header file

  GDouble par_A = GPU_UVARS(0);
  GDouble par_B = GPU_UVARS(1);
  GDouble par_C = GPU_UVARS(2);
  GDouble par_D = GPU_UVARS(3);
  GDouble par_E = GPU_UVARS(4);
  GDouble par_F = GPU_UVARS(5);
  GDouble par_G = GPU_UVARS(6);
  GDouble par_H = GPU_UVARS(7);
  GDouble par_L = GPU_UVARS(8);
    
  pcDevAmp[iEvent] = (1.0 + par_A*Y + par_B*Y*Y + par_C*X + par_D*X*X + par_E*X*Y + par_F*Y*Y*Y + par_G*X*X*Y + par_H*X*Y*Y + par_L*X*X*X);
}


void
DecayAmp_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,GDouble par_A, GDouble par_B, GDouble par_C, GDouble par_D, GDouble par_E, GDouble par_F, GDouble par_G, GDouble par_H, GDouble par_L)
{  

  DecayAmp_kernel<<< dimGrid, dimBlock >>>
    ( GPU_AMP_ARGS,par_A, par_B, par_C, par_D, par_E, par_F, par_G, par_H, par_L);

}
